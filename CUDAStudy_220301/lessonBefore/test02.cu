#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
 
using namespace std;
 
__device__ int add_one(int a) {
 
	return a + 1;
}
 
__global__ void show(int *a) {
 
	for (int i = 0; i < 10; i++) {
 
		//a[i] = add_one(a[i]);
		printf(" %d", a[i]);
	}
	printf("\n");
}
 
__global__ void changeValue(int *a) {
 
	for (int i = 0; i < 10; i++) {
 
		a[i] = 100;
	}
}
 
int main() {
 
	int cpu_int[10] = {10, 10, 10, 10, 10, 10, 10, 10, 10, 10};
	int *gpu_int;
 
	//在GPU上分配空间
	hipMalloc((void**)&gpu_int, 10 * sizeof(int));
	show << <1, 1 >> > (gpu_int);
 
	//初始化其值
	hipMemset(gpu_int, 0, 10 * sizeof(int));
	show<< <1, 1 >> > (gpu_int);
 
	//将cpu_int赋值给gpu_int
	hipMemcpy(gpu_int, cpu_int, 10 * sizeof(int), hipMemcpyHostToDevice);
	show << <1, 1 >> > (gpu_int);
 
	//改变gpu_int的值
	changeValue << <1, 1 >> >(gpu_int);
	show << <1, 1 >> > (gpu_int);
 
	//将gpu_int的值赋值到cpu_int
	hipMemcpy(cpu_int, gpu_int, 10 * sizeof(int), hipMemcpyDeviceToHost);
	printf("----------华丽的分割线----------\n");
	for (int i = 0; i < 10; i++) {
 
 
		printf(" %d", cpu_int[i]);
	}
 
 
	//释放gpu_int的空间
	hipFree(gpu_int);
 
 
	getchar();
	return 0;
}
