 
#include "hip/hip_runtime.h"
#include ""
 
#include <stdio.h>
 
 
int main(void) {
	hipDeviceProp_t  prop;  //将设备拥有的属性填充到hipDeviceProp_t结构中
 
	int count;
	hipGetDeviceCount(&count);  //支持CUDA架构的Device数量
	for (int i = 0; i< count; i++) {
		hipGetDeviceProperties(&prop, i);  //获取设备的属性
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
 
		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
 
		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n",
			prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n",
			prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n",
			prop.maxThreadsDim[0], prop.maxThreadsDim[1],
			prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n",
			prop.maxGridSize[0], prop.maxGridSize[1],
			prop.maxGridSize[2]);
		printf("\n");
	}
}
