#include "hip/hip_runtime.h"

#include <stdio.h>
// 接口函数： 主机代码调用GPU设备实现矢量加法 c = a + b
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
// 核函数：每个线程负责一个分量的加法
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x; // 获取线程ID
    c[i] = a[i] + b[i];
}
int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    // 并行矢量相加
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);
    // CUDA设备重置，以便其它性能检测和跟踪工具的运行，如Nsight and Visual Profiler to show complete traces.traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    return 0;
}
// 接口函数实现： 主机代码调用GPU设备实现矢量加法 c = a + b
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;
    // 选择程序运行在哪块GPU上，(多GPU机器可以选择)
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // 依次为 c = a + b三个矢量在GPU上开辟内存 .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // 将矢量a和b依次copy进入GPU内存中
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // 运行核函数，运行设置为1个block，每个block中size个线程
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
    // 检查是否出现了错误
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // 停止CPU端线程的执行，直到GPU完成之前CUDA的任务，包括kernel函数、数据拷贝等
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // 将计算结果从GPU复制到主机内存
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return cudaStatus;
}