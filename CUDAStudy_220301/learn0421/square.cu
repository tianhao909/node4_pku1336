#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void square(float* d_out,float* d_in){ // 下划线 gpu运行  square是kernel函数   
  int idx = threadIdx.x; 
  float f = d_in[idx]; 
  printf("f = %f \n",f);
  d_out[idx] = f * f; //求平方  算出d-out是在GPU上
}

int main(int argc,char** argv){ 
  const int ARRAY_SIZE = 8; //分配空间 定义数组size  8位数组
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float); 

  // generate the input array on the host
  float h_in[ARRAY_SIZE]; //h_开头的变量 表示 插在cpu上的变量    host  CPU编程 
  printf("===========\n"); 
  // ARRAY_SIZE=8  8个元素的数组  
  for(int i=0;i<ARRAY_SIZE;i++){  //给h_in[ARRAY_SIZE]数组的元素赋值
    printf("%d \n", i );

    //h_in[i] = float(i); // 类型转换
    h_in[i] = i; // 类型转换
    printf("============================\n"); 
    printf("h_in = %f \n", h_in[i]);
  }
  float h_out[ARRAY_SIZE]; 

  // declare GPU memory pointers
  float* d_in; //d_  表示GPU上的变量 device  GPU编程  指向GPU的地方 指针类型 指向float变量
  float* d_out; // 

  //要把 h_in数组 复制给d_in  
  // 最后是 d_out  复制给 h_out
  // allocate GPU memory  给GPU分配内存空间  
  hipMalloc((void**) &d_in,ARRAY_BYTES);   // 给d_in  和 d_out都分配了内存空间，空间大小是ARRAY_BYTES
  hipMalloc((void**) &d_out,ARRAY_BYTES);  //  d_out 传进去的是d_out的地址 ，  要修改d_out本身的值， 而不是d_out指向的值 ， 也不是指向d_out的地址
//把d_out看成普通变量， &d_out相当于是传递d_out的地址，而d_out是地址指针变量， 
//一个指针类型， 该类型的指针指向无类型指针
//https://zhuanlan.zhihu.com/p/88652961


  // transfer the array to GPU  第二步  cuda 内存复制 ，  
  hipMemcpy(d_in,h_in,ARRAY_BYTES,hipMemcpyHostToDevice); 
  // 复制的目标位置， 复制原位置 从host to device  复制量 

  // launch the kernel
  square<<<1,ARRAY_SIZE>>>(d_out,d_in);  //有1个线程块  有8个线程， 每个线程一起算
  //1 加载在多少线程块  创建爱你多少线程块  ARRAY_SIZE 每个线程块 多少线程
  //square的输入参数  __global__ void square(float* d_out,float* d_in)

  // copy back the result array to the GPU
  hipMemcpy(h_out,d_out,ARRAY_BYTES,hipMemcpyDeviceToHost); 
  // 目标host  原位置d  从device复制到host   把d-out从GPU复制到CPU的h-out中

  // print out the resulting array
  for(int i=0;i<ARRAY_SIZE;i++){
    printf("===============\n");
    // printf("%f",h_out[i]);
    printf("%f \n",h_out[i]);
    //printf(((i%4) != 3) ? "\t" : "\n");
  }

  // free GPU memory allocation
  hipFree(d_in); //释放GPU 和 CPU的
  hipFree(d_out);

  return 0;


}
